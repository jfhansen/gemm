/************************************
 * Baseline CUDA GeMM implementation
 * Author: jfhansen
 * Last modified: 15/07/2020
 ***********************************/


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstddef>
#include <assert.h>
#include <algorithm>
#include <random>
#include <math.h>

#define BLOCKSIZE 64

const int N = (1<<10);
const int SHMEM_SIZE = (1<<10);

// Kernel function that computes GeMM on CUDA threads
__global__
void cuda_gemm(const float *A, const float *B, float *C, size_t N)
{
	// Thread index across multiple blocks
	//const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t stride = blockDim.x;
	// Block index in grid
	const size_t col = blockIdx.x;
	// Thread index in block
	const size_t tid = threadIdx.x;

	// Shared memory: Each GPU block is responsible for dealing with 
	// the computation of at least one column of the C matrix.
	// Thereby a column of the B matrix can be brought into shared memory.
	// Since matrices are stored row-by-row, this saves a lot of time dealing
	// with cache misses.
	__shared__ float s_a[SHMEM_SIZE];
	__shared__ float s_b[SHMEM_SIZE];
	// Every thread in block fetches one element in column 'bid'.
	for (size_t idx = tid; idx < N; idx+=stride) {
		s_a[idx] = A[blockIdx.x*N+idx];
		s_b[idx] = B[idx*N+col];
	}
	__syncthreads();
	
	// Perform GeMM
	for (size_t row = tid; row < N; row+=stride)
		for (size_t j = 0; j < N; j++)
			C[row*N+col] += s_a[j] * s_b[j];
	//__syncthreads();
}

__global__
void count_zero_elems(const float *C, size_t N, float *nzelem)
{
	unsigned int stride = blockDim.x;
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// Block loads elements into shared memory
	__shared__ float y[SHMEM_SIZE];
	// If element is zero write 1 in y array
	// Each block takes care of 1 column of input array
	for (size_t row = tid; row < N; row+=stride)
		y[row] = (C[row*N+bid] == 0) ? 1.0 : 0.0;
	__syncthreads();
	
	// Build summation tree. Find amount of zero elements in each column
	for (int s=N/2; s>0; s=s/2)
	{
		for (size_t row = tid; row < s; row+=stride)
		{
			y[row] += y[row+s];
		}
		__syncthreads();
	}
	// Thread 0 in every block holds amount of zero elements in one column.
	if (tid == 0)
		atomicAdd(nzelem, y[tid]);
}

float verify_result(const float *A, const float *B, float *C, size_t N)
{
	unsigned count_v = 0;
	unsigned count_c = 0;
	float average_error = 0;
	float max_error = 0;
	for (size_t i = 0; i < N; i++)
	{
        for (size_t j = 0; j < N; j++)
		{
			float tmp = 0;
            for (size_t k = 0; k < N; k++)
			{
                tmp += A[i*N + k] * B[k*N + j];
			}
			if (tmp == 0)
				count_v++;
			if (C[i*N+j] == 0)
				count_c++;
			average_error += fabs(tmp - C[i*N+j]);
			max_error = fmax(max_error, tmp - C[i*N+j]);
			//assert(tmp == C[i*N+j]);
		}
	}
	average_error /= N*N;
	std::cout << "Average Error: " << average_error << std::endl;
	std::cout << "Number of zero elements in verification: " << count_v << std::endl;
	std::cout << "Number of zero elements in C: " << count_c << std::endl;
	return max_error;
}

int main() {
	uint32_t bytes = N*N*sizeof(float);	

	// Allocate memory in host.
	float *h_a, *h_b, *h_c;
	float *h_nzelem;
	h_a = new float[N*N];
	h_b = new float[N*N];
	h_c = new float[N*N];
	h_nzelem = new float;
	*h_nzelem = 0;
	
	// Allocate unified memory
	//cudaMallocManaged(&a, N*N*sizeof(float));
	//cudaMallocManaged(&b, N*N*sizeof(float));
	//cudaMallocManaged(&c, N*N*sizeof(float));

	// Generate values from uniform distribution
	std::mt19937 rng;
	rng.seed(std::random_device()());
	std::uniform_real_distribution<float> dist(-10,10);

	// Fill A and B matrices with random values
	// Initialize C as zero-only matrix
	std::generate(h_a, h_a+N*N, [&] { return dist(rng); });
	std::generate(h_b, h_b+N*N, [&] { return dist(rng); });
	std::generate(h_c, h_c+N*N, [&] { return 0.0; });

	// Allocate device memory
	float *d_a, *d_b, *d_c;
	float *d_nzelem;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);
	hipMalloc(&d_nzelem, sizeof(float));
	
	// Copy data to device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_nzelem, h_nzelem, sizeof(float), hipMemcpyHostToDevice);
	
	// Compute number of threads per block and number of blocks
	unsigned blockSize = BLOCKSIZE;
	unsigned numBlocks = N;
	// Verify that kernel works.
	count_zero_elems<<<numBlocks, blockSize>>>(d_c,N,d_nzelem);
	hipDeviceSynchronize();

	hipMemcpy(h_nzelem, d_nzelem, sizeof(float), hipMemcpyDeviceToHost);
	// Should be N*N
	std::cout << "Number of zero elements in C before: " << *h_nzelem << std::endl;
	*h_nzelem = 0;

	hipMemcpy(d_nzelem, h_nzelem, sizeof(float), hipMemcpyHostToDevice);
	
	// Run GeMM kernel on GPU
	cuda_gemm<<<numBlocks, blockSize>>>(d_a,d_b,d_c,N);
	// Wait for GPU to finish
	hipDeviceSynchronize();
	
	// Count number of zero elements in C matrix after GeMM
	count_zero_elems<<<numBlocks, blockSize>>>(d_c,N,d_nzelem);
	hipDeviceSynchronize();
	
	hipMemcpy(h_nzelem, d_nzelem, sizeof(float), hipMemcpyDeviceToHost);
	// Should be 0
	std::cout << "Number of zero elements in C after: " << *h_nzelem << std::endl;
	
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	// Verify result
	float maxError;
	maxError =	verify_result(h_a,h_b,h_c,N);
	std::cout << "Max Error: " << maxError << std::endl;
	
	// Free Memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	delete [] h_a;
	delete [] h_b;
	delete [] h_c;
	
	return 0;
}
