/************************************
 * Baseline CUDA integer GeMM implementation
 * Author: jfhansen
 * Last modified: 15/07/2020
 ***********************************/


#include <hip/hip_runtime.h>
#include <iostream>
#include <cstddef>
#include <assert.h>
#include <algorithm>
#include <random>
#include <math.h>

#define BLOCKSIZE 256

// Kernel function that computes GeMM on CUDA threads
__global__
void cuda_gemm(const int *A, const int *B, int *C, size_t N)
{
	// Block index in grid
	const size_t col = blockIdx.x * blockDim.x + threadIdx.x;
	// Thread index in block
	const size_t row = blockIdx.y * blockDim.y + threadIdx.y;

	// Perform GeMM
	for (size_t j = 0; j < N; j++)
		C[row*N+col] += A[row*N+j] * B[j*N+col];
}

__global__
void count_zero_elems(const int *C, size_t N, int *nzelem)
{
	unsigned int stride = blockDim.x;
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// Block loads elements into shared memory
	extern __shared__ int y[];
	// If element is zero write 1 in y array
	// Each block takes care of 1 column of input array
	for (size_t row = tid; row < N; row+=stride)
		y[row] = (C[row*N+bid] == 0) ? 1 : 0;
	__syncthreads();
	
	// Build summation tree. Find amount of zero elements in each column
	for (int s=N/2; s>0; s=s/2)
	{
		for (size_t row = tid; row < s; row+=stride)
		{
			y[row] += y[row+s];
		}
	}
	__syncthreads();
	// Thread 0 in every block holds amount of zero elements in one column.
	if (tid == 0)
		atomicAdd(nzelem, y[tid]);
	
	__syncthreads();
}

float verify_result(const int *A, const int *B, int *C, size_t N)
{
	unsigned count_v = 0;
	unsigned count_c = 0;
	float average_error = 0;
	float max_error = 0;
	for (size_t i = 0; i < N; i++)
	{
        for (size_t j = 0; j < N; j++)
		{
			float tmp = 0;
            for (size_t k = 0; k < N; k++)
			{
                tmp += A[i*N + k] * B[k*N + j];
			}
			if (tmp == 0)
				count_v++;
			if (C[i*N+j] == 0)
				count_c++;
			average_error += fabs(tmp - C[i*N+j]);
			max_error = fmax(max_error, tmp - C[i*N+j]);
			//assert(tmp == C[i*N+j]);
		}
	}
	average_error /= N*N;
	std::cout << "Average Error: " << average_error << std::endl;
	std::cout << "Number of zero elements in verification: " << count_v << std::endl;
	std::cout << "Number of zero elements in C: " << count_c << std::endl;
	return max_error;
}

int main() {
	int N = (1<<10);
	uint32_t bytes = N*N*sizeof(int);	

	// Allocate memory in host.
	int *h_a, *h_b, *h_c;
	int *h_nzelems;
	h_a = new int[N*N];
	h_b = new int[N*N];
	h_c = new int[N*N];
	h_nzelems = new int;
	*h_nzelems = 0;
	
	// Allocate unified memory
	//cudaMallocManaged(&a, N*N*sizeof(float));
	//cudaMallocManaged(&b, N*N*sizeof(float));
	//cudaMallocManaged(&c, N*N*sizeof(float));

	// Generate values from uniform distribution
	std::mt19937 rng;
	rng.seed(std::random_device()());
	std::uniform_int_distribution<int> dist(-10,10);

	// Fill A and B matrices with random values
	// Initialize C as zero-only matrix
	std::generate(h_a, h_a+N*N, [&] { return dist(rng); });
	std::generate(h_b, h_b+N*N, [&] { return dist(rng); });
	std::generate(h_c, h_c+N*N, [&] { return 0.0; });

	// Allocate device memory
	int *d_a, *d_b, *d_c;
	int *d_nzelems;
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);
	hipMalloc(&d_nzelems, sizeof(int));
	
	// Copy data to device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_nzelems, h_nzelems, sizeof(int), hipMemcpyHostToDevice);

	// Compute block size and number of blocks for GeMM
	unsigned THREADS = sqrt(BLOCKSIZE);
	unsigned BLOCKS = N/THREADS;
	dim3 threads(THREADS,THREADS);
	dim3 blocks(BLOCKS,BLOCKS);
	// Run GeMM kernel on GPU
	cuda_gemm<<<blocks, threads>>>(d_a,d_b,d_c,N);
	// Wait for GPU to finish
	hipDeviceSynchronize();
	
	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

	unsigned blockSize = BLOCKSIZE;
	unsigned numBlocks = N;
	count_zero_elems<<<numBlocks, blockSize, N*sizeof(int)>>>(d_c, N, d_nzelems);
	hipDeviceSynchronize();

	hipMemcpy(h_nzelems, d_nzelems, sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "Number of Zero elements in C: " << *h_nzelems << std::endl;

	// Verify result
	float maxError;
	maxError =	verify_result(h_a,h_b,h_c,N);
	std::cout << "Max Error: " << maxError << std::endl;
	
	// Free Memory
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	delete [] h_a;
	delete [] h_b;
	delete [] h_c;
	
	hipError_t err;
	while ( (err = hipGetLastError()) != hipSuccess)
		std::cout << "CUDA Error: " << hipGetErrorString(err) << std::endl;
	return 0;
}
